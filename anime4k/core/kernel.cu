#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "opencv2/opencv.hpp"

#include <stdio.h>
#include <algorithm>
#include <iostream>
#include <chrono>
#define max2(a,b) a>b?a:b
#define min2(a,b) a<b?a:b
#define max3(a,b,c) (a>b?a:b)>c?(a>b?a:b):c
#define min4(a,b,c,d) (a<b?a:b)<(c<d?c:d)?(a<b?a:b):(c<d?c:d)

texture<float, 2> tex_img_deriv;
texture<float, 2> tex_output_deriv;

__device__ void compare(float *point, bool &result) {
	//�ഫһ����ռ���һλ
	float max = max3(point[0], point[1], point[2]);
	float min = min4(point[3], point[4], point[5], point[6]);
	result = (max < min);
}
__device__ void condition(bool reverse, int choice, int idx, int idy, bool &result, int *point, int* h_gpu, int* w_gpu) {
	int xl = max2(-idx, -1);
	int xr = min2((*w_gpu) - 1 - idx, 1);
	int yu = max2(-idy, -1);
	int yd = min2((*h_gpu) - 1 - idy, 1);
	float surround[3][3];
	if (reverse) {
		surround[0][0] = tex2D(tex_output_deriv, idx + xl, idy + yu);
		surround[0][1] = tex2D(tex_output_deriv, idx, idy + yu);
		surround[0][2] = tex2D(tex_output_deriv, idx + xr, idy + yu);
		surround[1][0] = tex2D(tex_output_deriv, idx + xl, idy);
		surround[1][1] = tex2D(tex_output_deriv, idx, idy);
		surround[1][2] = tex2D(tex_output_deriv, idx + xr, idy);
		surround[2][0] = tex2D(tex_output_deriv, idx + xl, idy + yd);
		surround[2][1] = tex2D(tex_output_deriv, idx, idy + yd);
		surround[2][2] = tex2D(tex_output_deriv, idx + xr, idy + yd);
	}
	else {
		surround[0][0] = tex2D(tex_img_deriv, idx + xl, idy + yu);
		surround[0][1] = tex2D(tex_img_deriv, idx, idy + yu);
		surround[0][2] = tex2D(tex_img_deriv, idx + xr, idy + yu);
		surround[1][0] = tex2D(tex_img_deriv, idx + xl, idy);
		surround[1][1] = tex2D(tex_img_deriv, idx, idy);
		surround[1][2] = tex2D(tex_img_deriv, idx + xr, idy);
		surround[2][0] = tex2D(tex_img_deriv, idx + xl, idy + yd);
		surround[2][1] = tex2D(tex_img_deriv, idx, idy + yd);
		surround[2][2] = tex2D(tex_img_deriv, idx + xr, idy + yd);
	}
	if (choice == 0) {
		float input[] = { surround[0][0], surround[0][1], surround[0][2], surround[1][1], surround[2][0], surround[2][1], surround[2][2] };
		point[0] = idx + xl;
		point[1] = idy + yu;
		point[2] = idx;
		point[3] = idy + yu;
		point[4] = idx + xr;
		point[5] = idy + yu;
		compare(input, result);
	}
	else if (choice == 1) {
		float input[] = { surround[0][1], surround[0][2], surround[1][2], surround[1][0], surround[1][1], surround[2][1],255 };
		point[0] = idx;
		point[1] = idy + yu;
		point[2] = idx + xr;
		point[3] = idy + yu;
		point[4] = idx + xr;
		point[5] = idy;
		compare(input, result);
	}
	else if (choice == 2) {
		float input[] = { surround[0][2], surround[1][2], surround[2][2], surround[0][0], surround[1][0], surround[1][1], surround[2][0] };
		point[0] = idx + xr;
		point[1] = idy + yu;
		point[2] = idx + xr;
		point[3] = idy;
		point[4] = idx + xr;
		point[5] = idy + yd;
		compare(input, result);
	}
	else if (choice == 3) {
		float input[] = { surround[1][2], surround[2][2], surround[2][1], surround[0][1], surround[1][0], surround[1][1],255 };
		point[0] = idx + xr;
		point[1] = idy;
		point[2] = idx + xr;
		point[3] = idy + yd;
		point[4] = idx;
		point[5] = idy + yd;
		compare(input, result);
	}
	else if (choice == 4) {
		float input[] = { surround[2][0], surround[2][1], surround[2][2], surround[0][0], surround[0][1], surround[0][2], surround[1][1] };
		point[0] = idx + xr;
		point[1] = idy + yd;
		point[2] = idx;
		point[3] = idy + yd;
		point[4] = idx + xl;
		point[5] = idy + yd;
		compare(input, result);
	}
	else if (choice == 5) {
		float input[] = { surround[1][0], surround[2][0], surround[2][1], surround[0][1], surround[1][1], surround[1][2],255 };
		point[0] = idx + xl;
		point[1] = idy;
		point[2] = idx + xl;
		point[3] = idy + yd;
		point[4] = idx;
		point[5] = idy + yd;
		compare(input, result);
	}
	else if (choice == 6) {
		float input[] = { surround[0][0], surround[1][0], surround[2][0], surround[0][2], surround[1][1], surround[1][2], surround[2][2] };
		point[0] = idx + xl;
		point[1] = idy + yd;
		point[2] = idx + xl;
		point[3] = idy;
		point[4] = idx + xl;
		point[5] = idy + yu;
		compare(input, result);
	}
	else if (choice == 7) {
		float input[] = { surround[0][0], surround[0][1], surround[1][0], surround[1][1], surround[1][2], surround[2][1] ,255 };
		point[0] = idx + xl;
		point[1] = idy;
		point[2] = idx + xl;
		point[3] = idy + yu;
		point[4] = idx;
		point[5] = idy + yu;
		compare(input, result);
	}
	else {
		result = false;
	}
}
__global__ void deblur(float *src, float *dst, float *dst_deriv, int *nbytes_gpu, int* h_gpu, int* w_gpu, bool reverse) {
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	int idy = blockIdx.y*blockDim.y + threadIdx.y;
	int index = idy*(*w_gpu) + idx;
	while (idx<(*w_gpu) && idy<(*h_gpu)) {
		float alpha = 175.0f / 255;
		bool result = false;
		int point[] = { 0,0,0,0,0,0 };
		for (int i = 0; i < 8; ++i) {
			condition(reverse, i, idx, idy, result, point, h_gpu, w_gpu);
			if (result == false) {
				float new_color_b = 0;
				float new_color_g = 0;
				float new_color_r = 0;
				float new_luminance = 0;
				float mean_b = 0;
				float mean_g = 0;
				float mean_r = 0;
				mean_b = (src[3 * (point[0] + point[1] * (*w_gpu))] + src[3 * (point[2] + point[3] * (*w_gpu))] + src[3 * (point[4] + point[5] * (*w_gpu))]) / 3;
				mean_g = (src[3 * (point[0] + point[1] * (*w_gpu)) + 1] + src[3 * (point[2] + point[3] * (*w_gpu)) + 1] + src[3 * (point[4] + point[5] * (*w_gpu)) + 1]) / 3;
				mean_r = (src[3 * (point[0] + point[1] * (*w_gpu)) + 2] + src[3 * (point[2] + point[3] * (*w_gpu)) + 2] + src[3 * (point[4] + point[5] * (*w_gpu)) + 2]) / 3;
				new_color_b = alpha*src[3 * index] + (1 - alpha)*mean_b;
				new_color_g = alpha*src[3 * index + 1] + (1 - alpha)*mean_g;
				new_color_r = alpha*src[3 * index + 2] + (1 - alpha)*mean_r;
				new_luminance = 1. / 6 * new_color_b + 1. / 2 * new_color_g + 1. / 3 * new_color_r;
				if (dst_deriv[index] < new_luminance) {
					dst_deriv[index] = new_luminance;
					dst[3 * index] = new_color_b;
					dst[3 * index + 1] = new_color_g;
					dst[3 * index + 2] = new_color_r;
				}
			}
		}
		idx += blockDim.x*gridDim.x;
		if (idx >= (*w_gpu)) {
			idx = idx % (*w_gpu);
			idy+= blockDim.y*gridDim.y;
		}
	}
}
__global__ void calculate_grad(bool reverse, float *dst_deriv, int *nbytes_gpu, int* h_gpu, int* w_gpu) {
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	int idy = blockIdx.y* blockDim.y + threadIdx.y;
	int index = idy*(*w_gpu) + idx;
	while (idx<(*w_gpu) && idy<(*h_gpu)) {
		int xl = max2(-idx, -1);
		int xr = min2((*w_gpu) - 1 - idx, 1);
		int yu = max2(-idy, -1);
		int yd = min2((*h_gpu) - 1 - idy, 1);
		if (reverse) {
			dst_deriv[index] = abs(-tex2D(tex_output_deriv, idx + xl, idy + yu) - 2 * tex2D(tex_output_deriv, idx, idy + yu) - tex2D(tex_output_deriv, idx + xr, idy + yu) + tex2D(tex_output_deriv, idx + xl, idy + yd) + 2 * tex2D(tex_output_deriv, idx, idy + yd) + tex2D(tex_output_deriv, idx + xr, idy + yd))
				+ abs(-tex2D(tex_output_deriv, idx + xl, idy + yu) - 2 * tex2D(tex_output_deriv, idx + xl, idy) - tex2D(tex_output_deriv, idx + xl, idy + yd) + tex2D(tex_output_deriv, idx + xr, idy + yd) + 2 * tex2D(tex_output_deriv, idx + xr, idy) + tex2D(tex_output_deriv, idx + xr, idy + yu));

		}
		else {
			dst_deriv[index] = abs(-tex2D(tex_img_deriv, idx + xl, idy + yu) - 2 * tex2D(tex_img_deriv, idx, idy + yu) - tex2D(tex_img_deriv, idx + xr, idy + yu) + tex2D(tex_img_deriv, idx + xl, idy + yd) + 2 * tex2D(tex_img_deriv, idx, idy + yd) + tex2D(tex_img_deriv, idx + xr, idy + yd))
				+ abs(-tex2D(tex_img_deriv, idx + xl, idy + yu) - 2 * tex2D(tex_img_deriv, idx + xl, idy) - tex2D(tex_img_deriv, idx + xl, idy + yd) + tex2D(tex_img_deriv, idx + xr, idy + yd) + 2 * tex2D(tex_img_deriv, idx + xr, idy) + tex2D(tex_img_deriv, idx + xr, idy + yu));

		}
		dst_deriv[index] = min2(max2(dst_deriv[index] / 2, 0), 255);
		idx += blockDim.x*gridDim.x;
		if (idx >= (*w_gpu)) {
			idx = idx % (*w_gpu);
			idy += blockDim.y*gridDim.y;
		}
	}
}
__global__ void grad_refine(float *src, float *dst, float *dst_deriv, int *nbytes_gpu, int* h_gpu, int* w_gpu, bool reverse) {
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	int idy = blockIdx.y* blockDim.y + threadIdx.y;
	int index = idy*(*w_gpu) + idx;
	while (idx<(*w_gpu) && idy<(*h_gpu)) {
		float alpha = 0.0f / 255;
		bool result = false;
		int point[] = { 0,0,0,0,0,0 };
		if (reverse) {
			dst_deriv[index] = tex2D(tex_output_deriv, idx, idy);
		}
		else {
			dst_deriv[index] = tex2D(tex_img_deriv, idx, idy);
		}
		dst[3 * index] = src[3 * index];
		dst[3 * index + 1] = src[3 * index + 1];
		dst[3 * index + 2] = src[3 * index + 2];
		for (int i = 0; i < 8; ++i) {
			condition(reverse, i, idx, idy, result, point, h_gpu, w_gpu);
			if (result == true) {
				float new_color_b = 0;
				float new_color_g = 0;
				float new_color_r = 0;
				float deriv = 0;
				float mean_b = 0;
				float mean_g = 0;
				float mean_r = 0;
				mean_b = (src[3 * (point[0] + point[1] * (*w_gpu))] + src[3 * (point[2] + point[3] * (*w_gpu))] + src[3 * (point[4] + point[5] * (*w_gpu))]) / 3;
				mean_g = (src[3 * (point[0] + point[1] * (*w_gpu)) + 1] + src[3 * (point[2] + point[3] * (*w_gpu)) + 1] + src[3 * (point[4] + point[5] * (*w_gpu)) + 1]) / 3;
				mean_r = (src[3 * (point[0] + point[1] * (*w_gpu)) + 2] + src[3 * (point[2] + point[3] * (*w_gpu)) + 2] + src[3 * (point[4] + point[5] * (*w_gpu)) + 2]) / 3;
				new_color_b = alpha*src[3 * index] + (1 - alpha)*mean_b;
				new_color_g = alpha*src[3 * index + 1] + (1 - alpha)*mean_g;
				new_color_r = alpha*src[3 * index + 2] + (1 - alpha)*mean_r;
				if (reverse) {
					deriv = alpha*tex2D(tex_output_deriv, idx, idy) + (1 - alpha)*(tex2D(tex_output_deriv, point[0], point[1]) + tex2D(tex_output_deriv, point[2], point[3]) + tex2D(tex_output_deriv, point[4], point[5])) / 3;
				}
				else {
					deriv = alpha*tex2D(tex_img_deriv, idx, idy) + (1 - alpha)*(tex2D(tex_img_deriv, point[0], point[1]) + tex2D(tex_img_deriv, point[2], point[3]) + tex2D(tex_img_deriv, point[4], point[5])) / 3;
				}
				dst_deriv[index] = deriv;
				dst[3 * index] = new_color_b;
				dst[3 * index + 1] = new_color_g;
				dst[3 * index + 2] = new_color_r;
				break;
			}
		}
		idx += blockDim.x*gridDim.x;
		if (idx >= (*w_gpu)) {
			idx = idx % (*w_gpu);
			idy += blockDim.y*gridDim.y;
		}
	}
}

__global__ void img_luminance(float *src, float *dst, int* h_gpu, int* w_gpu) {
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	int idy = blockIdx.y* blockDim.y + threadIdx.y;
	int index = idy*(*w_gpu) + idx;
	while (idx<(*w_gpu) && idy<(*h_gpu)) {
		dst[index] = (src[3 * index] + 3 * src[3 * index + 1] + 2 * src[3 * index + 2]) / 6;
		idx += blockDim.x*gridDim.x;
		if (idx >= (*w_gpu)) {
			idx = idx % (*w_gpu);
			idy += blockDim.y*gridDim.y;
		}
	}
}
int main(int argc, char **argv)
{
	std::string img_path = argv[1];
	std::string save_path = argv[2];
	std::cout<<img_path<<std::endl;
	std::cout<<save_path<<std::endl;
	cv::Mat img = cv::imread(img_path);
	assert(img.channels() == 3);
	cv::resize(img, img, cv::Size((int)img.cols * 2, (int)img.rows * 2), 0, 0, cv::INTER_CUBIC);
	int w = img.cols;
	int h = img.rows;
	int nbytes = w*h * sizeof(float);
	dim3 thread_num(64, 64);
	dim3 block_num(16, 16);
	//dim3 grid = (w*h + block.x - 1) / block.x;
	//float *img_vector = new float[3 * nbytes / sizeof(float)];
	float *img_vector;
	hipHostMalloc((void**)&img_vector, 3 * nbytes, hipHostMallocDefault);
	for (int i = 0; i < h; ++i) {
		for (int j = 0; j < w; ++j) {
			cv::Vec3b* in_row = img.ptr<cv::Vec3b>(i);
			img_vector[3 * (i*w + j)] = in_row[j][0];
			img_vector[3 * (i*w + j) + 1] = in_row[j][1];
			img_vector[3 * (i*w + j) + 2] = in_row[j][2];
		}
	}

	//���ȴ�ӡ����channel*col
	//Ȼ��row��ӡ
	//ת��һά���� ����ֱ�Ӱ����˳���ν�����
	float *input, *output, *img_deriv, *output_deriv;
	int *w_gpu, *h_gpu, *nbytes_gpu;
	hipMalloc((float**)&input, 3 * nbytes);
	hipMalloc((float**)&output, 3 * nbytes);
	hipMalloc((float**)&img_deriv, nbytes);
	hipMalloc((float**)&output_deriv, nbytes);
	hipMalloc((int**)&w_gpu, sizeof(int));
	hipMalloc((int**)&h_gpu, sizeof(int));
	hipMalloc((int**)&nbytes_gpu, sizeof(int));
	hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
	hipBindTexture2D(NULL, tex_img_deriv, img_deriv, desc, w, h, nbytes);
	hipBindTexture2D(NULL, tex_output_deriv, output_deriv, desc, w, h, nbytes);
	float *resul;
	hipHostMalloc((void**)&resul, 3 * nbytes, hipHostMallocDefault);
	//float *result_deriv = new float[nbytes / sizeof(float)];
	hipEvent_t g_start, g_end;
	hipEventCreate(&g_start, 0);
	hipEventCreate(&g_end, 0);
	float time;
	std::chrono::high_resolution_clock::time_point start = std::chrono::high_resolution_clock::now();

		hipMemcpy(input, img_vector, 3 * nbytes, hipMemcpyHostToDevice);
		hipMemcpy(w_gpu, &w, sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(h_gpu, &h, sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(nbytes_gpu, &nbytes, sizeof(int), hipMemcpyHostToDevice);
    while (1) {
		hipEventRecord(g_start);
		img_luminance << <block_num, thread_num >> > (input, img_deriv, h_gpu, w_gpu);

		deblur << <block_num, thread_num >> > (input, output, output_deriv, nbytes_gpu, h_gpu, w_gpu, false);

		calculate_grad << <block_num, thread_num >> > (true, img_deriv, nbytes_gpu, h_gpu, w_gpu);
		/*
		hipMemcpy(result_deriv, img_deriv, nbytes, hipMemcpyDeviceToHost);
		cv::Mat grad_pic = cv::Mat::zeros(img.size(), CV_8UC1);
		for (int i = 0; i < h; ++i) {
		for (int j = 0; j < w; ++j) {
		uchar* out_row = grad_pic.ptr<uchar>(i);
		out_row[j] = (int)result_deriv[(i*w + j)];
		}
		}
		cv::imwrite("grad.bmp", grad_pic);
		*/
		grad_refine << <block_num, thread_num >> > (output, input, output_deriv, nbytes_gpu, h_gpu, w_gpu, false);
		hipMemcpy(resul, input, 3 * nbytes, hipMemcpyDeviceToHost);
		hipEventRecord(g_end);
		hipEventSynchronize(g_end);
		hipEventElapsedTime(&time, g_start, g_end);
		std::cout << "GPU time: " << time << std::endl;
	}
		
	//hipMemcpy(result_deriv, output_deriv, nbytes, hipMemcpyDeviceToHost);
	/*
	float *test = new float[3 * nbytes / sizeof(float)];
	hipMemcpy(test, output, 3 * nbytes, hipMemcpyDeviceToHost);
	cv::Mat test_pic = cv::Mat::zeros(img.size(), CV_8UC3);
	for (int i = 0; i < h; ++i) {
	for (int j = 0; j < w; ++j) {
	cv::Vec3b* out_row = test_pic.ptr<cv::Vec3b>(i);
	out_row[j][0] = (int)test[3 * (i*w + j)];
	out_row[j][1] = (int)test[3 * (i*w + j) + 1];
	out_row[j][2] = (int)test[3 * (i*w + j) + 2];
	}
	}
	cv::imwrite("test.bmp", test_pic);
	*/

	cv::Mat out_pic = cv::Mat::zeros(img.size(), img.type());
	for (int i = 0; i < h; ++i) {
		for (int j = 0; j < w; ++j) {
			cv::Vec3b* out_row = out_pic.ptr<cv::Vec3b>(i);
			out_row[j][0] = (int)resul[3 * (i*w + j)];
			out_row[j][1] = (int)resul[3 * (i*w + j) + 1];
			out_row[j][2] = (int)resul[3 * (i*w + j) + 2];
		}
	}

	cv::imwrite(save_path, out_pic);
	std::chrono::duration<double, std::milli> time_span = std::chrono::high_resolution_clock::now() - start;
	std::cout << "time:" << time_span.count() << std::endl;

	hipUnbindTexture(tex_img_deriv);
	hipUnbindTexture(tex_output_deriv);
	hipEventDestroy(g_start);
	hipEventDestroy(g_end);
	hipHostFree(img_vector);
	hipHostFree(resul);
	hipFree(input);
	hipFree(output);
	hipFree(img_deriv);
	hipFree(output_deriv);
	hipFree(w_gpu);
	hipFree(h_gpu);
	hipFree(nbytes_gpu);

	return 0;
}

